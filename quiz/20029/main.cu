
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <assert.h>
#define DEBUG
#define UINT uint32_t
#define MAXN 1024
#define MAXGPU 2
#define MAX_TESTCASE 512
#define THREADS_PER_BLOCK 32
// in-place transpose
__global__ void cudaTranspose(int N, UINT target[][MAXN]){
    // Share works among row index
    // row index of target matrix
    int index_x = blockDim.x * blockIdx.x + threadIdx.x;
    UINT temp;
    // swap until diagonal index reaches
    for(int col = 0; col < index_x; col++){
        temp = target[index_x][col];
        target[index_x][col] = target[col][index_x];
        target[col][index_x] = temp;
    }
}
// multiple src1 and src2
// TODO: Fix out of range multiplication error
__global__ void cudaMultiply(int N, UINT src1[][MAXN], UINT src2[][MAXN], UINT target[][MAXN]){
    UINT sum = 0;
    __shared__ UINT local_src1[THREADS_PER_BLOCK][THREADS_PER_BLOCK];
    __shared__ UINT local_src2[THREADS_PER_BLOCK][THREADS_PER_BLOCK];
    // loop over each block
    for(int i = 0; i < gridDim.x; i++){
        // TODO: local transpose
        if(blockIdx.x * blockDim.x + threadIdx.x < N && i * blockDim.y + threadIdx.y < N){
            local_src1[threadIdx.x][threadIdx.y] = src1[blockIdx.x * THREADS_PER_BLOCK + threadIdx.x][i * THREADS_PER_BLOCK + threadIdx.y];
        }else{
            // padding
            local_src1[threadIdx.x][threadIdx.y] = 0;
        }
        if(i * blockDim.x + threadIdx.x < N && blockIdx.y * blockDim.y + threadIdx.y < N){
            local_src2[threadIdx.x][threadIdx.y] = src2[i * THREADS_PER_BLOCK + threadIdx.x][blockIdx.y * THREADS_PER_BLOCK + threadIdx.y];
        }else{
            //padding
            local_src2[threadIdx.x][threadIdx.y] = 0;
        }
        __syncthreads();
        // local matrix multiplication
        for(int j = 0; j < THREADS_PER_BLOCK; j++){
            // TODO: Cache miss(can optimize by transposing matrix above)
            sum += local_src1[threadIdx.x][j] * local_src2[j][threadIdx.y];
        }
        // Move to next block 
        __syncthreads();
    }
    target[blockIdx.x * blockDim.x + threadIdx.x][blockIdx.y * blockDim.y + threadIdx.y] = sum;
}
__global__ void cudaAdd(int N, UINT src1[][MAXN], UINT src2[][MAXN], UINT target[][MAXN]){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    target[x][y] = src1[x][y] + src2[x][y];
}

void rand_gen(UINT c, int N, UINT A[][MAXN]) {
    UINT x = 2, n = N*N;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            x = (x * x + c + i + j)%n;
            A[i][j] = x;
        }
    }
}
void print_matrix(int N, UINT A[][MAXN]) {
    for (int i = 0; i < N; i++) {
        fprintf(stderr, "[");
        for (int j = 0; j < N; j++)
            fprintf(stderr, " %u", A[i][j]);
        fprintf(stderr, " ]\n");
    }
}
UINT signature(int N, UINT A[][MAXN]) {
    UINT h = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            h = (h + A[i][j]) * 2654435761LU;
    }
    return h;
}
void device2host(UINT cudaMem[][MAXN], UINT hostMem[][MAXN]){
    hipError_t error;
    error = hipMemcpy(hostMem, cudaMem, sizeof(UINT)*MAXN*MAXN, hipMemcpyDeviceToHost);
    if(error == hipErrorInvalidValue){
        fprintf(stderr, "cudaErrorInvalidValue\n");
    }else if(error == hipErrorInvalidDevicePointer){
        fprintf(stderr, "cudaErrorInvalidDevicePointer\n");
    }else if(error == hipErrorInvalidMemcpyDirection){
        fprintf(stderr, "cudaErrorInvalidMemcpyDirection\n");
    }
    assert(error == hipSuccess);
}
void host2device(UINT cudaMem[][MAXN], UINT hostMem[][MAXN]){
    hipError_t error;
    error = hipMemcpy(cudaMem[0], hostMem[0], sizeof(UINT)*MAXN*MAXN, hipMemcpyHostToDevice);
    assert(error == hipSuccess);
}

UINT signatures[MAX_TESTCASE];
UINT X[MAXGPU][MAXN][MAXN], Y[MAXGPU][MAXN][MAXN];
UINT A[MAXGPU][MAXN][MAXN], B[MAXGPU][MAXN][MAXN];
UINT (*cudaX[MAXGPU])[MAXN], (*cudaY[MAXGPU])[MAXN];
UINT (*cudaA[MAXGPU])[MAXN], (*cudaB[MAXGPU])[MAXN];
UINT (*cudaAB[MAXGPU])[MAXN], (*cudaBA[MAXGPU])[MAXN];
UINT (*cudaABA[MAXGPU])[MAXN], (*cudaBAB[MAXGPU])[MAXN];
int main() {
    int N[MAX_TESTCASE], S[MAX_TESTCASE][2]; // TODO: May have bug because S_i <= 2^31
    int currentGPU = 0;
    int testcase_num = 0;
    hipError_t ret;
    // allocate memory to each GPU device
    for(int i = 0; i < MAXGPU; i++){
        ret = hipMalloc(&cudaX[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
        ret = hipMalloc(&cudaY[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
        ret = hipMalloc(&cudaA[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
        ret = hipMalloc(&cudaB[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
        ret = hipMalloc(&cudaAB[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
        ret = hipMalloc(&cudaBA[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
        ret = hipMalloc(&cudaABA[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
        ret = hipMalloc(&cudaBAB[i], sizeof(UINT) * MAXN * MAXN);
        assert(ret == hipSuccess);
    }
    for(testcase_num = 0; scanf("%d", &N[testcase_num]) == 1; testcase_num++){
        scanf("%d", &S[testcase_num][0]);
        scanf("%d", &S[testcase_num][1]);
    }
    assert(testcase_num <= MAX_TESTCASE);
    // Multi-GPU load balancing
    for(int i = 0; i < testcase_num; i++){
        currentGPU = i % MAXGPU;
        ret = hipSetDevice(currentGPU);
        assert(ret == hipSuccess);
        int n = N[i], s_a = S[i][0], s_b = S[i][1];
        rand_gen(s_a, n, A[currentGPU]);
        rand_gen(s_b, n, B[currentGPU]);
        host2device(cudaA[currentGPU], A[currentGPU]);
        host2device(cudaB[currentGPU], B[currentGPU]);
#ifdef DEBUG
        memset(A[currentGPU], 0, sizeof(UINT) * MAXN * MAXN);
        memset(B[currentGPU], 0, sizeof(UINT) * MAXN * MAXN);
        device2host(cudaA[currentGPU], A[currentGPU]);
        device2host(cudaB[currentGPU], B[currentGPU]);
        printf("Matrix A:\n");
        print_matrix(n, A[currentGPU]);
        printf("Matrix B:\n");
        print_matrix(n, B[currentGPU]);
#endif
        int blocksPerGrid = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        int threadsPerBlock = THREADS_PER_BLOCK;
        dim3 dimBlock(threadsPerBlock, threadsPerBlock);
        dim3 dimGrid(blocksPerGrid, blocksPerGrid);
        // AB
        cudaMultiply<<< dimGrid, dimBlock >>>(n, cudaA[currentGPU], cudaB[currentGPU], cudaAB[currentGPU]);
        // BA
        cudaMultiply<<< dimGrid, dimBlock >>>(n, cudaB[currentGPU], cudaA[currentGPU], cudaBA[currentGPU]);
        // AB+BA
        hipDeviceSynchronize(); // make sure AB, BA are done
        cudaAdd<<< dimGrid, dimBlock >>>(n, cudaAB[currentGPU], cudaBA[currentGPU], cudaX[currentGPU]);
        // ABA
        cudaMultiply<<< dimGrid, dimBlock >>>(n, cudaAB[currentGPU], cudaA[currentGPU], cudaABA[currentGPU]);
        // BAB
        cudaMultiply<<< dimGrid, dimBlock >>>(n, cudaBA[currentGPU], cudaB[currentGPU], cudaBAB[currentGPU]);
        // ABA+BAB
        hipDeviceSynchronize();
        cudaAdd<<< dimGrid, dimBlock >>>(n, cudaABA[currentGPU], cudaBAB[currentGPU], cudaY[currentGPU]);
        // [*] Print results
        device2host(cudaX[currentGPU], X[currentGPU]);
        printf("%u\n", signature(n, X[currentGPU]));
        device2host(cudaY[currentGPU], Y[currentGPU]);
        printf("%u\n", signature(n, Y[currentGPU]));
    }
    for(int i = 0; i < MAXGPU; i++){
        hipFree(cudaX[i]);
        hipFree(cudaY[i]);
        hipFree(cudaA[i]);
        hipFree(cudaB[i]);
        hipFree(cudaAB[i]);
        hipFree(cudaBA[i]);
        hipFree(cudaABA[i]);
        hipFree(cudaBAB[i]);
    }
    return 0;
}
